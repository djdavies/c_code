#include "hip/hip_runtime.h"
// This example demonstrates how to check for CUDA errors,
// which indicate programming bugs, or otherwise broken
// program behavior.

#include <stdlib.h>
#include <stdio.h>

// The "fill-with-7" kernel
__global__ void kernel(int *array) {
  array[blockIdx.x * blockDim.x + threadIdx.x] = 7;
}

int main (int argc, char **argv) {
  int num_elements = 256;                      // Arrays of 256 elements
  int num_bytes = num_elements * sizeof(int);  // Size of the arrays in bytes

  // Pointers to host & device arrays
  int *device_array = 0;
  int *host_array = 0;

  // malloc a host array
  host_array = (int*)malloc(num_bytes);
  // hipMalloc a device array
  hipMalloc((void**)&device_array, num_bytes);
  // If either memory allocation failed, report an error message
  if(host_array == 0 || device_array == 0) {
    printf("couldn't allocate memory\n");
    return 1;
  }

  // Choose a grid configuration
  int block_size = 128;
  int grid_size = num_elements / block_size;

  // Accidentally pass the kernel a pointer to the host array:
  // this will cause the kernel to crash when it attempts to
  // dereferences a host pointer
  kernel<<<grid_size,block_size>>> (host_array);

  // Make the host wait until the kernel is finished executing before
  // checking for the last CUDA error. Otherwise, we won't detect
  // an error if one occurred.
  hipDeviceSynchronize();

  // This kind of "blocking" operation is usually only appropriate during
  // the debugging process because it forces the host to wait on the device
  // while it could be busy doing other things. once the code has been
  // debugged, frequent error checking code should be eliminated or disabled.

  // Ask CUDA for the last error to occur (if one exists)
  hipError_t error = hipGetLastError();
  if(error != hipSuccess) {
    // Something's gone wrong: print out the CUDA error as a string
    printf("CUDA Error: %s\n", hipGetErrorString(error));
    // We can't recover from the error -- exit the program
    return 1;
  }

  // No error occurred, proceed as usual

  // Download and inspect the result on the host:
  hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);

  // Print out the first 10 elements
  for(int i=0; i < 10; ++i) {
    printf("%d ", host_array[i]);
  }
  printf("\n");

  // Deallocate memory
  free(host_array);
  hipFree(device_array);

  return 0;
}
